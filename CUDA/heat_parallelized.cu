
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// Simple define to index into a 1D array from 2D space
#define I2D(num, c, r) ((r)*(num)+(c))

/*
 * `step_kernel_mod` is currently a direct copy of the CPU reference solution
 * `step_kernel_ref` below. Accelerate it to run as a CUDA kernel.
 */

__global__ void step_kernel_mod(int ni, int nj, float fact, float* temp_in, float* temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i>0 && i<ni-1){
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (j>0 && j<nj-1){
      // find indices into linear memory
      // for central point and neighbours
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i-1, j);
      ip10 = I2D(ni, i+1, j);
      i0m1 = I2D(ni, i, j-1);
      i0p1 = I2D(ni, i, j+1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10]-2*temp_in[i00]+temp_in[ip10];
      d2tdy2 = temp_in[i0m1]-2*temp_in[i00]+temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00]+fact*(d2tdx2 + d2tdy2);
    }
  }
}

void step_kernel_ref(int ni, int nj, float fact, float* temp_in, float* temp_out)
{
  int i00, im10, ip10, i0m1, i0p1;
  float d2tdx2, d2tdy2;


  // loop over all points in domain (except boundary)
  for ( int j=1; j < nj-1; j++ ) {
    for ( int i=1; i < ni-1; i++ ) {
      // find indices into linear memory
      // for central point and neighbours
      i00 = I2D(ni, i, j);
      im10 = I2D(ni, i-1, j);
      ip10 = I2D(ni, i+1, j);
      i0m1 = I2D(ni, i, j-1);
      i0p1 = I2D(ni, i, j+1);

      // evaluate derivatives
      d2tdx2 = temp_in[im10]-2*temp_in[i00]+temp_in[ip10];
      d2tdy2 = temp_in[i0m1]-2*temp_in[i00]+temp_in[i0p1];

      // update temperatures
      temp_out[i00] = temp_in[i00]+fact*(d2tdx2 + d2tdy2);
    }
  }
}

int main()
{
  int istep;
  int nstep = 200; // number of time steps

  // Specify our 2D dimensions
  const int ni = 1000;
  const int nj = 1000;
  float tfac = 8.418e-5; // thermal diffusivity of silver

  float *temp1_ref, *temp2_ref, *temp1, *temp2, *temp_tmp, *dev_temp1, *dev_temp2;

  const int size = ni * nj * sizeof(float);

  temp1_ref = (float*)malloc(size);
  temp2_ref = (float*)malloc(size);
  temp1 = (float*)malloc(size);
  temp2 = (float*)malloc(size);
  hipMalloc((void**)&dev_temp1, ni * nj * sizeof(float));
  hipMalloc((void**)&dev_temp2, ni * nj * sizeof(float));
  // Initialize with random data
  for( int i = 0; i < ni*nj; ++i) {
    temp1_ref[i] = temp2_ref[i] = temp1[i] = temp2[i] = (float)rand()/(float)(RAND_MAX/100.0f);
  }
  
  // Execute the CPU-only reference version
  for (istep=0; istep < nstep; istep++) {
    step_kernel_ref(ni, nj, tfac, temp1_ref, temp2_ref);

    // swap the temperature pointers
    temp_tmp = temp1_ref;
    temp1_ref = temp2_ref;
    temp2_ref= temp_tmp;
  }
   	
  // Execute the modified version using same data
  dim3 n_threads_per_block(32,32); //32x32=1024
  dim3 n_blocks(ni/32+1,nj/32+1);
  
  clock_t start = clock();
  
  for (istep=0; istep < nstep; istep++) {
    hipMemcpy(dev_temp1, temp1, ni * nj *sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_temp2, temp2, ni * nj *sizeof(float), hipMemcpyHostToDevice);
    step_kernel_mod<<<n_blocks,n_threads_per_block>>>(ni, nj, tfac, dev_temp1, dev_temp2);
    hipMemcpy(temp1, dev_temp1, ni * nj * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(temp2, dev_temp2, ni * nj * sizeof(float), hipMemcpyDeviceToHost);
    // swap the temperature pointers
    temp_tmp = temp1;
    temp1 = temp2;
    temp2= temp_tmp; 
  }
  hipDeviceSynchronize();
  clock_t end = clock();
  
  float maxError = 0;
  // Output should always be stored in the temp1 and temp1_ref at this point
  for( int i = 0; i < ni*nj; ++i ) {
    if (abs(temp1[i]-temp1_ref[i]) > maxError) { maxError = abs(temp1[i]-temp1_ref[i]); }
  }

  // Check and see if our maxError is greater than an error bound
  if (maxError > 0.0005f)
    printf("Problem! The Max Error of %.5f is NOT within acceptable bounds.\n", maxError);
  else{
    printf("The Max Error of %.5f is within acceptable bounds.\n", maxError);
	printf("Elapsed time: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
	}

  hipFree( dev_temp1 );
  hipFree( dev_temp2 );
  free( temp1_ref );
  free( temp2_ref );
  free( temp1 );
  free( temp2 );
  return 0;
}
